#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "nbody.h"

/*
  1. *i - index of particle to be compute_forced
  2. *nparticles - # of particles
  3. d_p points to the whole particles 
*/

__device__ double atomicAddDouble(double* address, double val)
{
  unsigned long long int* address_as_ull =
  (unsigned long long int*) address;
  unsigned long long int old = *address_as_ull;
  unsigned long long int assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
    __double_as_longlong(val + __longlong_as_double(assumed)));
    // Note: uses integer comparison to avoid hang in case
    // of NaN (since NaN != NaN)
  } while (assumed != old);
  return __longlong_as_double(old);
}

__global__ void __compute_force__ (int * i, int * nparticles, particle_t * d_p) 
{
  
  particle_t * computed_p = &d_p[*i];  
  int j;

  // __syncthreads();

  for(j = blockIdx.x * blockDim.x + threadIdx.x;
      j < *nparticles;
      j += blockDim.x*gridDim.x) 
  {
    particle_t * p = &d_p[j];
    // change the cpu version to cuda version
    double x_sep, y_sep, dist_sq, grav_base;

    x_sep = p->x_pos - computed_p->x_pos;
    y_sep = p->y_pos - computed_p->y_pos;
    dist_sq = MAX((x_sep*x_sep) + (y_sep*y_sep), 0.01);

    /* Use the 2-dimensional gravity rule: F = d * (GMm/d^2) */
    grav_base = GRAV_CONSTANT * (computed_p->mass) * (p->mass)/dist_sq;

    // computed_p->x_force += grav_base*x_sep;
    // computed_p->y_force += grav_base*y_sep;

    // using atomicAdd
    atomicAddDouble(&(computed_p->x_force), grav_base*x_sep);
    atomicAddDouble(&(computed_p->y_force), grav_base*y_sep);

  }

}

extern "C" void cuda_compute_force(int i, int nparticles, particle_t * p)
{

  particle_t * d_p;
  int * d_i;
  int * d_nparticles;

  // allocate space for device copies
  hipMalloc((void **)&d_p, nparticles * sizeof(particle_t));
  hipMalloc((void **)&d_i, sizeof(int));
  hipMalloc((void **)&d_nparticles, sizeof(int)); 

  hipError_t cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    printf(hipGetErrorString(cudaStatus));
  }
  
  // copy inputs to device
  hipMemcpy(d_p, p, nparticles * sizeof(particle_t), hipMemcpyHostToDevice);
  hipMemcpy(d_i, &i, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_nparticles, &nparticles, sizeof(int), hipMemcpyHostToDevice);

  cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    printf(hipGetErrorString(cudaStatus));
  }
  
  // TODO: check the return .
  int thr_per_blk = 1024; // maximum
  int blk_in_grid = (int) ceil( float(nparticles) / thr_per_blk );

  __compute_force__<<<blk_in_grid,thr_per_blk>>>(d_i, d_nparticles, d_p);
  // __compute_force__<<<1,2>>>(d_i, d_nparticles, d_p);
  hipDeviceSynchronize();

  // copy result back to host
  hipMemcpy(&p[i], &d_p[i], sizeof(particle_t), hipMemcpyDeviceToHost);

  // cleanup
  hipFree(d_p);
  hipFree(d_i);
  hipFree(d_nparticles);

}